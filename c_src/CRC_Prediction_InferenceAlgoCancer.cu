#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "CRC_Prediction_InferenceAlgoCancer.h"

using namespace std;

#define swap(a, b) {float *hold = a; a = b; b = hold;}
#define swapD(a, b) {double *hold = a; a = b; b = hold;}
#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif
#define	kBlockSize	32
#define	kBlockRounding	31
#define	kBigBlockSize	1024
#define	kBigBlockRounding	1023

hipblasHandle_t *handles = NULL;


/**
 * Matrix multiplication and addition on the device: {@code result = (mulMatrix * fxnMatrix) + addMatrix}
 * {@code widthMul} is {@code mulMatrix}'s width and {@code widthFxn} is {@code fxnMatrix}'s width
 * {@code result} and {@code addMatrix} have height {@code heightMul and width {@code widthFxn}
 */
template <int BLOCK_SIZE, class T> __global__ void
matrixMulCUDA (T *result, T *mulMatrix, T *fxnMatrix, T *addMatrix, int widthMul, int heightMul, int widthFxn, int heightFxn)
{
	// Block index
	int		blockX = blockIdx.x * BLOCK_SIZE;
	int		blockY = blockIdx.y * BLOCK_SIZE;
	
	// Thread index
	int		threadX = threadIdx.x;
	int		threadY = threadIdx.y;
	int		maxFxn = widthFxn * heightFxn;
	int		maxMul = widthMul * heightMul;
	bool	deadFxnX, deadMulY;
	bool	deadResX = deadFxnX = (blockX + threadX) >= widthFxn;	// Are these within the valid range of the results matrix?
	bool	deadResY = deadMulY = (blockY + threadY) >= heightMul;
	
	// Offsets from upper left corner to the cells of interest for this thread
	int	mulOffset = (widthMul * threadY) + threadX;
	int	fxnOffset = (widthFxn * threadY) + threadX;
	
	// Index of the first sub-matrix of mulMatrix processed by the block
	int	mulBegin = widthMul * blockY;
	
	// Index of the first sub-matrix of mulMatrix NOT processed by the block
	int	mulEnd = mulBegin + widthMul;
	
	// Step size used to iterate through the sub-matrices of mulMatrix
	int	mulStep = BLOCK_SIZE;
	
	// Index of the first sub-matrix of fxnMatrix processed by the block
	int	fxnBegin = blockX;
	
	// Step size used to iterate through the sub-matrices of fxnMatrix
	int	fxnStep = BLOCK_SIZE * widthFxn;
	
	// resultSub is used to store the element of the block sub-matrix
	// that is computed by the thread
	T	resultSub = 0;
	
	// Loop over all the sub-matrices of mulMatrix and fxnMatrix
	// required to compute the block sub-matrix
	for (int mulUpperLeft = mulBegin, fxnUpperLeft = fxnBegin; mulUpperLeft < mulEnd; mulUpperLeft += mulStep, fxnUpperLeft += fxnStep)
	{
		__shared__ T	mulSub[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ T	fxnSub[BLOCK_SIZE][BLOCK_SIZE];
		
//		printf ("Loading data: Block[%d, %d]: Thread[%d, %d]\n", blockX, blockY, threadX, threadY);
		
		// Load the matrices from device memory to shared memory; each thread loads
		// one element of each matrix.	If block size doesn't evenly divide into array 
		// size, will be attempting to read past the edge.	Make those 0, so nothing 
		// extra gets added
		int mulPos = mulUpperLeft + mulOffset;
		if (!deadMulY && (mulPos < maxMul))
			mulSub[threadY][threadX] = mulMatrix[mulPos];
		else
			mulSub[threadY][threadX] = 0.0f;
		
		int fxnPos = fxnUpperLeft + fxnOffset;
		if (!deadFxnX && (fxnPos < maxFxn))
			fxnSub[threadY][threadX] = fxnMatrix[fxnPos];
		else
			fxnSub[threadY][threadX] = 0.0f;
		
		// Synchronize to make sure the matrices are loaded
		__syncthreads ();
		
//		printf ("Loaded data: Block[%d, %d]: Thread[%d, %d]\n", blockX, blockY, threadX, threadY);
		
		// Multiply the two matrices together; each thread computes one element
		// of the block sub-matrix
#pragma unroll
		
		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			resultSub += mulSub[threadY][k] * fxnSub[k][threadX];
		}
		
		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of mulMatrix and fxnMatrix in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory; each thread writes one element
	if (!deadResX && !deadResY)
	{
		int	pos = ((blockY + threadY) * widthFxn) + blockX + threadX;	// (numRows * width) + numCols
		result[pos] = resultSub + addMatrix[pos];
	}
}


/**
 * Compare the contents of two T[].  If any value of {@code first} differs from {@code second} 
 * by {@code epsilon} or more, return false.  Else return true
 * 
 * @param first		double[] holding values to test, same length as {@code second}
 * @param second	double[] holding values to test, same length as {@code first}
 * @param size		Length of both {@code first} and {@code second}
 * @param epsilon	Test value.  All matching elements of {@code first} and {@code second} must 
 * differ by less than this
 * @param diffCount	Count of values in {@code first} that differ from {@code second} by {@code epsilon} or more, 
 */
template <int BLOCK_SIZE, class T> __global__ void
compare (T *first, T *second, size_t size, T epsilon, size_t *diffCount)
{
	size_t	blockX = blockIdx.x * BLOCK_SIZE;
	size_t	threadX = threadIdx.x;
	size_t	pos = blockX + threadX;
	
	if (pos < size)
	{
		T	value = first[pos] - second[pos];
		
		if (value < 0)
			value = -value;
		
		if (value > epsilon)
			++(*diffCount);
	}
}


/**
 * Copy the contents of {@code source} into {@code target}
 * 
 * @param source	float[] / double[] to read from.  Must not be null, and length {@code theSize}
 * @param target	float[] / double[] to write to.  Must not be null, and length {@code theSize}
 * @param theSize	Size of both arrays
 */
template <int BLOCK_SIZE, class T> __global__ void
copy (const T *source, T *target, int theSize)
{
	// Block index
	int	blockX = blockIdx.x * BLOCK_SIZE;
	
	// Thread index
	int	threadX = threadIdx.x;
	
	//  Source location
	int	pos = blockX + threadX;
	
	if (pos < theSize)
		target[pos] = source[pos];
}


/**
 * Write the transpose of {@code source}, which is in column:row order (i.e. laid out with the first 
 * column being in positions 0 - {@code numRows - 1} into {@code target} in row:column order
 * 
 * @param source	float[] / double[] to read from.  Must not be null, and length {@code numRows * numCols}
 * @param target	float[] / double[] to write to.  Must not be null, and length {@code numRows * numCols}
 * @param numRows	Number of rows in {@code source}, will be number of cols in {@code target}
 * @param numCols	Number of cols in {@code source}, will be number of rows in {@code target}
 */
template <int BLOCK_SIZE, class T, class U> __global__ void
transposeCR (const T *source, U *target, int numRows, int numCols)
{
	// Block index
	int	blockX = blockIdx.x * BLOCK_SIZE;
	int	blockY = blockIdx.y * BLOCK_SIZE;
	
	// Thread index
	int	threadX = threadIdx.x;
	int	threadY = threadIdx.y;
	
	//  Source location
	int	row = blockY + threadY;
	int	col = blockX + threadX;
	
	if ((row < numRows) && (col < numCols))
	{
		int	readPos = (col * numRows) + row;
		int	writePos = (row * numCols) + col;
		
		target[writePos] = (U) (source[readPos]);
	}
}


/**
 * Write the transpose of {@code source}, which is in row:column order (i.e. laid out with the first 
 * row being in positions 0 - {@code numCols - 1} into {@code target} in column:row order
 * 
 * @param source	float[] / double[] to read from.  Must not be null, and length {@code numRows * numCols}
 * @param target	float[] / double[] to write to.  Must not be null, and length {@code numRows * numCols}
 * @param numRows	Number of rows in {@code source}, will be number of cols in {@code target}
 * @param numCols	Number of cols in {@code source}, will be number of rows in {@code target}
 */
template <int BLOCK_SIZE, class T, class U> __global__ void
transposeRC (const T *source, U *target, int numRows, int numCols)
{
	// Block index
	int	blockX = blockIdx.x * BLOCK_SIZE;
	int	blockY = blockIdx.y * BLOCK_SIZE;
	
	// Thread index
	int	threadX = threadIdx.x;
	int	threadY = threadIdx.y;
	
	//  Source location
	int	row = blockY + threadY;
	int	col = blockX + threadX;
	
	if ((row < numRows) && (col < numCols))
	{
		int	readPos = (row * numCols) + col;
		int	writePos = (col * numRows) + row;
		
		target[writePos] = (U) (source[readPos]);
	}
}


/**
 * Write the transpose of {@code source} into {@code target}, on the Host
 * 
 * @param source	float / double[] to read from.  Must not be null, and length {@code numRows * numCols}
 * @param target	float / double[] to write to.  Must not be null, and length {@code numRows * numCols}
 * @param numRows	Number of rows in {@code source}, will be number of cols in {@code target}
 * @param numCols	Number of cols in {@code source}, will be number of rows in {@code target}
 */
template <class T, class U> void transposeRCHost (const T source[], U target[], int numRows, int numCols)
{
	int	i, readPos = 0;
	
	for (i = 0; i < numRows; ++i)
	{
		int	j, writePos = i;
		
		for (j = 0; j < numCols; ++j)
		{
			target[writePos] = (U) (source[readPos]);
			++readPos;
			writePos += numRows;
		}
	}
}


/**
 * Write the transpose of {@code source} into {@code target}
 * 
 * @param source	float / double[] to read from.  Must not be null, and length {@code numRows * numCols}
 * @param target	float / double[] to write to.  Must not be null, and length {@code numRows * numCols}
 * @param numRows	Number of rows in {@code source}, will be number of cols in {@code target}
 * @param numCols	Number of cols in {@code source}, will be number of rows in {@code target}
 */
template <class T> void transpose (T source[], T target[], int numRows, int numCols)
{
	int	i, readPos = 0;
	
	for (i = 0; i < numRows; ++i)
	{
		int	j, writePos = i;
		
		for (j = 0; j < numCols; ++j)
		{
			target[writePos] = source[readPos];
			++readPos;
			writePos += numRows;
		}
	}
}


/**
 *	Chose which GPU to use, exiting on any error
 */
void cSetDevice (int whichGPU, int line)
{
	hipError_t	error = hipSetDevice (whichGPU);

	if (error != hipSuccess)
	{
		printf ("hipSetDevice (%d) returned error %s (code %d), line (%d)\n", whichGPU, hipGetErrorString (error), error, line);
		exit (EXIT_FAILURE);
	}
}


/**
 *	Chose which GPU to use, exiting on any error
 */
void cuBlasSetup (hipblasHandle_t *handle, int whichGPU, int line)
{
	cSetDevice (whichGPU, line);
	
	hipblasStatus_t	stat = hipblasCreate (handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("CUBLAS initialization failed, hipblasCreate () returned error %s (code %d), line (%d)\n", _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 * Matrix multiplication and addition on the device: {@code result = (mulMatrix * fxnMatrix) + addMatrix}
 * {@code result} and {@code addMatrix} have height {@code rowsMul} and width {@code colsFxn}
 *
 * @param handle	{@link hipblasHandle_t} that says what CuBLAS settings to use
 * @param result	{@code float *} to fill in with results of processing.  Current contents ignored
 * @param mulMatrix	{@code float *} A array for AB + C matrix multiplication and addition
 * @param fxnMatrix	{@code float *} B array for AB + C matrix multiplication and addition
 * @param addMatrix	{@code float *} C array for AB + C matrix multiplication and addition
 * @param colsMul	{@code mulMatrix}'s width, must equal {@code rowsFxn}
 * @param rowsMul	{@code mulMatrix}'s height, also height of {@code result} and {@code addMatrix}
 * @param colsFxn	{@code fxnMatrix}'s width, also width of {@code result} and {@code addMatrix}
 * @param rowsFxn	{@code fxnMatrix}'s height, must equal {@code colsMul}
 * @param line		Line routine was called from, used when reporting errors
 * @param iteration	Iteration calling routine was on when called this, used when reporting errors
 */
void cuBlasMul (hipblasHandle_t handle, float *result, float *mulMatrix, float *fxnMatrix, float *addMatrix, 
				int colsMul, int rowsMul, int colsFxn, int rowsFxn, int line, int iteration)
{
	hipblasStatus_t	stat;
	const float		alpha = 1.0f;
	const float		beta = 1.0f;
	int				arraySize = colsFxn * rowsFxn;
	
	// Replace contents of result with addMatrix, so can use the beta add, rather than a separate operation
//	printf ("Calling copy\n");
	copy<kBigBlockSize, float><<< (arraySize + kBigBlockRounding) / kBigBlockSize, kBigBlockSize >>>(addMatrix, result, arraySize);
//	printf ("Calling hipDeviceSynchronize\n");
	hipDeviceSynchronize ();
	
//	printf ("Calling hipblasSgemm\n");
	stat = hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsMul, colsFxn, rowsFxn, &alpha, 
						mulMatrix, rowsMul, fxnMatrix, rowsFxn, &beta, result, rowsMul);
//	printf ("testing return code\n");
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("hipblasSgemm () iteration: %d, returned error %s (code %d), call line (%d)\n", iteration, _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 * Matrix multiplication and addition on the device: {@code result = (mulMatrix * fxnMatrix) + addMatrix}
 * {@code result} and {@code addMatrix} have height {@code rowsMul} and width {@code colsFxn}
 *
 * @param handle	{@link hipblasHandle_t} that says what CuBLAS settings to use
 * @param result	{@code double *} to fill in with results of processing.  Current contents ignored
 * @param mulMatrix	{@code double *} A array for AB + C matrix multiplication and addition
 * @param fxnMatrix	{@code double *} B array for AB + C matrix multiplication and addition
 * @param addMatrix	{@code double *} C array for AB + C matrix multiplication and addition
 * @param colsMul	{@code mulMatrix}'s width, must equal {@code rowsFxn}
 * @param rowsMul	{@code mulMatrix}'s height, also height of {@code result} and {@code addMatrix}
 * @param colsFxn	{@code fxnMatrix}'s width, also width of {@code result} and {@code addMatrix}
 * @param rowsFxn	{@code fxnMatrix}'s height, must equal {@code colsMul}
 * @param line		Line routine was called from, used when reporting errors
 * @param iteration	Iteration calling routine was on when called this, used when reporting errors
 */
void cuBlasMul (hipblasHandle_t handle, double *result, double *mulMatrix, double *fxnMatrix, double *addMatrix, 
				int colsMul, int rowsMul, int colsFxn, int rowsFxn, int line, int iteration)
{
	hipblasStatus_t	stat;
	const double	alpha = 1.0f;
	const double	beta = 1.0f;
	int				arraySize = colsFxn * rowsFxn;
	
	// Replace contents of result with addMatrix, so can use the beta add, rather than a separate operation
	copy<kBigBlockSize, double><<< (arraySize + kBigBlockRounding) / kBigBlockSize, kBigBlockSize >>>(addMatrix, result, arraySize);
	hipDeviceSynchronize ();
	
	stat = hipblasDgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsMul, colsFxn, rowsFxn, &alpha, 
						mulMatrix, rowsMul, fxnMatrix, rowsFxn, &beta, result, rowsMul);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("hipblasDgemm () iteration: %d, returned error %s (code %d), call line (%d)\n", iteration, _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 * Matrix multiplication on the device: {@code result = firstMatrix * secondMatrix}
 * {@code result} has height {@code rowsFirst} and width {@code colsSecond}
 *
 * @param handle		{@link hipblasHandle_t} that says what CuBLAS settings to use
 * @param result		{@code double *} to fill in with results of processing.  Current contents ignored
 * @param firstMatrix	{@code double *} A array for AB matrix multiplication
 * @param secondMatrix	{@code double *} B array for AB matrix multiplication
 * @param colsFirst		{@code firstMatrix}'s width, must equal {@code rowsSecond}
 * @param rowsFirst		{@code firstMatrix}'s height, also height of {@code result}
 * @param colsSecond	{@code secondMatrix}'s width, also width of {@code result}
 * @param rowsSecond	{@code secondMatrix}'s height, must equal {@code colsFirst}
 * @param line			Line routine was called from, used when reporting errors
 */
void cuBlasMul (hipblasHandle_t handle, double *result, double *firstMatrix, double *secondMatrix, 
				int colsFirst, int rowsFirst, int colsSecond, int rowsSecond, int line)
{
	hipblasStatus_t	stat;
	const double	alpha = 1.0f;
	const double	beta = 0.0f;
	
	stat = hipblasDgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colsFirst, rowsSecond, colsSecond, &alpha, 
						firstMatrix, colsFirst, secondMatrix, colsSecond, &beta, result, colsFirst);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("hipblasDgemm () returned error %s (code %d), call line (%d)\n", _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 * Matrix multiplication on the device: {@code result = firstMatrix * secondMatrix}<br>
 * <b>Arrays are in C stype row major order, unlike all other calls</b>
 * {@code result} has height {@code rowsFirst} and width {@code colsSecond}
 *
 * @param handle		{@link hipblasHandle_t} that says what CuBLAS settings to use
 * @param result		{@code double *} to fill in with results of processing.  Current contents ignored
 * @param firstMatrix	{@code double *} A array for AB matrix multiplication
 * @param secondMatrix	{@code double *} B array for AB matrix multiplication
 * @param colsFirst		{@code firstMatrix}'s width, must equal {@code rowsSecond}
 * @param rowsFirst		{@code firstMatrix}'s height, also height of {@code result}
 * @param colsSecond	{@code secondMatrix}'s width, also width of {@code result}
 * @param rowsSecond	{@code secondMatrix}'s height, must equal {@code colsFirst}
 * @param line			Line routine was called from, used when reporting errors
 */
void cuBlasMulC (hipblasHandle_t handle, double *result, double *firstMatrix, double *secondMatrix, 
				 int colsFirst, int rowsFirst, int colsSecond, int rowsSecond, int line)
{
	hipblasStatus_t	stat;
	const double	alpha = 1.0f;
	const double	beta = 0.0f;
	
	stat = hipblasDgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colsSecond, rowsFirst, colsFirst, &alpha, 
						secondMatrix, colsSecond, firstMatrix, colsFirst, &beta, result, colsSecond);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("hipblasDgemm () returned error %s (code %d), call line (%d)\n", _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 *	Allocate cuda memory, exiting on any error
 */
void cMalloc (void **target, size_t theSize, const char *name, int line)
{
	hipError_t	error = hipMalloc (target, theSize);

	if (error != hipSuccess)
	{
		printf ("hipMalloc %s returned error %s (code %d), line (%d)\n", name, hipGetErrorString (error), error, line);
		exit (EXIT_FAILURE);
	}
}


/**
 *	Copy to or from cuda memory, exiting on any error
 */
void cCopy (void *target, const void *source, size_t theSize, enum hipMemcpyKind kind, const char *name, int line)
{
	hipError_t	error = hipMemcpy (target, source, theSize, kind);

	if (error != hipSuccess)
	{
		printf ("hipMemcpy %s returned error %s (code %d), line (%d)\n", name, hipGetErrorString (error), error, line);
		exit (EXIT_FAILURE);
	}
}


/**
 *	Routine to let us know the library really is loaded
 */
void initCuda (int numGPUs)
{
	handles = (hipblasHandle_t *) malloc (numGPUs * sizeof(hipblasHandle_t));
	
	for (int whichGPU = 0; whichGPU < numGPUs; ++whichGPU)
	{
		cuBlasSetup (handles + whichGPU, whichGPU, __LINE__);	// Create a hipblasHandle_t for each GPU
	}
	
	printf ("Finished initCuda (%d)\n", numGPUs);
}


/**
 *	Routine to let us know the library really is loaded
 */
JNIEXPORT void JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_initCuda (JNIEnv *env, jclass clazz, jint numGPUsJ)
{
	int	numGPUs = (int) numGPUsJ;
	
	initCuda (numGPUs);
}


/**
 * Cuda processing, using single precision for speed<br>
 * Process {@code fxnMatrix} {@code maximumIterations} times, or until no element changes as much 
 * as {@code epsilon}<br>
 * Processing: {@code (mulMatrix x fxnMatrix) + addMatrix}<br>
 * All matrices are laid out with the first col being in positions 0 - {@code numRows - 1}, 
 * <b>this is the layout of the {@link DoubleMatrix} data, and different from all the other 
 * JNI versions of this code
 * 
 * @param fxnMatrixJ	Key matrix
 * @param mulMatrixJ	Square matrix with same number of rows and columns as {@code fxnMatrix} has rows
 * @param addMatrixJ	Matrix with same dimensions as {@code fxnMatrix}
 * @param numRowsJ	Number of rows in {@code fxnMatrix} and {@code addMatrix}.<br>
 * Number of rows and columns in {@code mulMatrix}
 * @param numColsJ	Number of cols in {@code fxnMatrix} and {@code addMatrix}
 * @param maxIterations	Maximum number of iterations to run if don't converge
 * @param epsilonJ	Result is converged if every element of {@code fxnMatrix} changes by less than 
 * {@code epsilon} during one round of processing
 * @param whichGPUJ	Which GPU to target
 * @return	The transpose of the final matrix, whether or not it converged, as a 1-d matrix
 */
JNIEXPORT void JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_multiplyMatrixCuBLASD (
	JNIEnv *env, jclass clazz, jdoubleArray firstMatrixJ, jdoubleArray secondMatrixJ, 
	jdoubleArray resultsJ, jint resultRowsJ, jint resultColsJ, jint sharedDimJ, jint whichGPUJ)
{
	int	whichGPU = (int) whichGPUJ;
	
	if (handles == NULL)
		initCuda (max(4, whichGPU + 1));
	
	hipblasHandle_t	handle = handles[whichGPU];
	cSetDevice (whichGPU, __LINE__);
	
	int		resultRows = (int) resultRowsJ;
	int		numCols = (int) resultColsJ;
	int		sharedDim = (int) sharedDimJ;
	double	*firstMatrixH = env->GetDoubleArrayElements (firstMatrixJ, NULL);
	double	*secondMatrixH = env->GetDoubleArrayElements (secondMatrixJ, NULL);
	size_t	resultSize = resultRows * numCols;
	size_t	firstSize = resultRows * sharedDim;
	size_t	secondSize = sharedDim * numCols;
	size_t	memSizeFirst = firstSize * sizeof(double);
	size_t	memSizeSecond = secondSize * sizeof(double);
	size_t	memSizeResult = resultSize * sizeof(double);
	double	*firstMatrix, *secondMatrix, *resultHold;
	
//	struct timeval startTV;
//	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &firstMatrix, memSizeFirst, "firstMatrix", __LINE__);
	cMalloc ((void **) &secondMatrix, memSizeSecond, "secondMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeResult, "resultHold", __LINE__);
	
	// copy host memory to device
	cCopy (firstMatrix, firstMatrixH, memSizeFirst, hipMemcpyHostToDevice, "(firstMatrix, firstMatrixH)", __LINE__);
	cCopy (secondMatrix, secondMatrixH, memSizeSecond, hipMemcpyHostToDevice, "(secondMatrix, secondMatrixH)", __LINE__);
	
	cuBlasMulC (handle, resultHold, firstMatrix, secondMatrix, sharedDim, resultRows, numCols, sharedDim, __LINE__);
	hipDeviceSynchronize ();
	
//	struct timeval endTV;
//	gettimeofday (&endTV, NULL);
//	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
//	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
//	printf ("cuBlasMulC (double) used %lu microseconds for %d rows, %d cols, and %d sharedDim\n", done - theStart, resultRows, numCols, sharedDim);
	
	double	*results = env->GetDoubleArrayElements (resultsJ, NULL);
	cCopy (results, resultHold, memSizeResult, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	env->ReleaseDoubleArrayElements (resultsJ, results, 0);	// Copy these results back
	
	hipFree (firstMatrix);
	hipFree (secondMatrix);
	hipFree (resultHold);
	env->ReleaseDoubleArrayElements (secondMatrixJ, secondMatrixH, JNI_ABORT);
	env->ReleaseDoubleArrayElements (firstMatrixJ, firstMatrixH, JNI_ABORT);
}


/**
 * Cuda processing, using single precision for speed<br>
 * Process {@code fxnMatrix} {@code maximumIterations} times, or until no element changes as much 
 * as {@code epsilon}<br>
 * Processing: {@code (mulMatrix x fxnMatrix) + addMatrix}<br>
 * All matrices are laid out with the first col being in positions 0 - {@code numRows - 1}, 
 * <b>this is the layout of the {@link DoubleMatrix} data, and different from all the other 
 * JNI versions of this code
 * 
 * @param fxnMatrixJ	Key matrix
 * @param mulMatrixJ	Square matrix with same number of rows and columns as {@code fxnMatrix} has rows
 * @param addMatrixJ	Matrix with same dimensions as {@code fxnMatrix}
 * @param numRowsJ	Number of rows in {@code fxnMatrix} and {@code addMatrix}.<br>
 * Number of rows and columns in {@code mulMatrix}
 * @param numColsJ	Number of cols in {@code fxnMatrix} and {@code addMatrix}
 * @param maxIterations	Maximum number of iterations to run if don't converge
 * @param epsilonJ	Result is converged if every element of {@code fxnMatrix} changes by less than 
 * {@code epsilon} during one round of processing
 * @param whichGPUJ	Which GPU to target
 * @return	The transpose of the final matrix, whether or not it converged, as a 1-d matrix
 */
JNIEXPORT jboolean JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_convergeMatrixCuBLAS (
	JNIEnv *env, jclass clazz, jfloatArray fxnMatrixJ, jfloatArray mulMatrixJ, jfloatArray addMatrixJ, 
	jfloatArray resultsJ, jint numRowsJ, jint numColsJ, jint maxIterations, jint whichGPUJ, jfloat epsilonJ)
{
	int	whichGPU = (int) whichGPUJ;
	
	if (handles == NULL)
		initCuda (max(4, whichGPU + 1));
	
	hipblasHandle_t	handle = handles[whichGPU];
	cSetDevice (whichGPU, __LINE__);
	
	int		numRows = (int) numRowsJ;
	int		numCols = (int) numColsJ;
	int		maxIter = (int) maxIterations;
	float	epsilon = (float) epsilonJ;
	float	*fxnMatrixH = env->GetFloatArrayElements (fxnMatrixJ, NULL);
	float	*mulMatrixH = env->GetFloatArrayElements (mulMatrixJ, NULL);
	float	*addMatrixH = env->GetFloatArrayElements (addMatrixJ, NULL);
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(float);
	size_t	memSizeMul = numRows * numRows * sizeof(float);
	float	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
	size_t	changeCountH;
	size_t	*changeCount;
	jboolean	converged = false;
	
#ifdef DoPrint
	printf ("In convergeMatrixCudaCuBLAS, numRows: %d, numCols: %d, maxIter: %d\n", numRows, numCols, maxIter);
	printArray ("mulMatrix", mulMatrixH, numRows * numRows);
	printArray ("fxnMatrix", fxnMatrixH, arraySize);
	printArray ("addMatrix", addMatrixH, arraySize);
#endif
	
//	struct timeval startTV;
//	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	
	int		i, numBlocks = (arraySize + kBigBlockRounding) / kBigBlockSize;
	for (i = 0; i < maxIter; ++i)
	{
#ifdef DoPrint
		printArray ("first", first, arraySize);
#endif
		cuBlasMul (handle, resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows, __LINE__, i);
		hipDeviceSynchronize ();
		swap(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<kBigBlockSize, float><<< numBlocks, kBigBlockSize >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
	}
	
//	struct timeval endTV;
//	gettimeofday (&endTV, NULL);
//	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
//	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
//	printf ("cuBlasMul (float) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	float	*results = env->GetFloatArrayElements (resultsJ, NULL);
	cCopy (results, fxnMatrix, memSizeFxn, hipMemcpyDeviceToHost, "(results, result)", __LINE__);
//	cCopy (fxnMatrixH, results, memSizeFxn, hipMemcpyDeviceToHost, "(results, result)", __LINE__);
	env->ReleaseFloatArrayElements (resultsJ, results, 0);	// Copy these results back
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	env->ReleaseFloatArrayElements (fxnMatrixJ, fxnMatrixH, JNI_ABORT);
	env->ReleaseFloatArrayElements (mulMatrixJ, mulMatrixH, JNI_ABORT);
	env->ReleaseFloatArrayElements (addMatrixJ, addMatrixH, JNI_ABORT);
	
	return converged;
}


/**
 * Cuda processing, using single precision for speed<br>
 * Process {@code fxnMatrix} {@code maximumIterations} times, or until no element changes as much 
 * as {@code epsilon}<br>
 * Processing: {@code (mulMatrix x fxnMatrix) + addMatrix}<br>
 * All matrices are laid out with the first col being in positions 0 - {@code numRows - 1}, 
 * <b>this is the layout of the {@link DoubleMatrix} data, and different from all the other 
 * JNI versions of this code
 * 
 * @param fxnMatrixJ	Key matrix
 * @param mulMatrixJ	Square matrix with same number of rows and columns as {@code fxnMatrix} has rows
 * @param addMatrixJ	Matrix with same dimensions as {@code fxnMatrix}
 * @param numRowsJ	Number of rows in {@code fxnMatrix} and {@code addMatrix}.<br>
 * Number of rows and columns in {@code mulMatrix}
 * @param numColsJ	Number of cols in {@code fxnMatrix} and {@code addMatrix}
 * @param maxIterations	Maximum number of iterations to run if don't converge
 * @param epsilonJ	Result is converged if every element of {@code fxnMatrix} changes by less than 
 * {@code epsilon} during one round of processing
 * @param whichGPUJ	Which GPU to target
 * @return	The transpose of the final matrix, whether or not it converged, as a 1-d matrix
 */
JNIEXPORT jboolean JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_convergeMatrixCuBLASD (
	JNIEnv *env, jclass clazz, jdoubleArray fxnMatrixJ, jdoubleArray mulMatrixJ, jdoubleArray addMatrixJ, 
	jdoubleArray resultsJ, jint numRowsJ, jint numColsJ, jint maxIterations, jint whichGPUJ, jdouble epsilonJ)
{
	int	whichGPU = (int) whichGPUJ;
	
	if (handles == NULL)
		initCuda (max(4, whichGPU + 1));
	
	hipblasHandle_t	handle = handles[whichGPU];
	cSetDevice (whichGPU, __LINE__);
	
	int		numRows = (int) numRowsJ;
	int		numCols = (int) numColsJ;
	int		maxIter = (int) maxIterations;
	double	epsilon = (double) epsilonJ;
	double	*fxnMatrixH = env->GetDoubleArrayElements (fxnMatrixJ, NULL);
	double	*mulMatrixH = env->GetDoubleArrayElements (mulMatrixJ, NULL);
	double	*addMatrixH = env->GetDoubleArrayElements (addMatrixJ, NULL);
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(double);
	size_t	memSizeMul = numRows * numRows * sizeof(double);
	double	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
	size_t	changeCountH;
	size_t	*changeCount;
	jboolean	converged = false;
	
#ifdef DoPrint
	printf ("In convergeMatrixCudaCuBLASD, numRows: %d, numCols: %d, maxIter: %d\n", numRows, numCols, maxIter);
	printArray ("mulMatrix", mulMatrixH, numRows * numRows);
	printArray ("fxnMatrix", fxnMatrixH, arraySize);
	printArray ("addMatrix", addMatrixH, arraySize);
#endif
	
//	struct timeval startTV;
//	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	int	i, numBlocks = (arraySize + kBigBlockRounding) / kBigBlockSize;
	
	for (i = 0; i < maxIter; ++i)
	{
		cuBlasMul (handle, resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows, __LINE__, i);
		hipDeviceSynchronize ();
		swapD(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<kBigBlockSize, double><<< numBlocks, kBigBlockSize >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
	}
	
//	struct timeval endTV;
//	gettimeofday (&endTV, NULL);
//	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
//	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
//	printf ("cuBlasMul (double) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	double	*results = env->GetDoubleArrayElements (resultsJ, NULL);
	cCopy (results, fxnMatrix, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	env->ReleaseDoubleArrayElements (resultsJ, results, 0);	// Copy these results back
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	env->ReleaseDoubleArrayElements (fxnMatrixJ, fxnMatrixH, JNI_ABORT);
	env->ReleaseDoubleArrayElements (mulMatrixJ, mulMatrixH, JNI_ABORT);
	env->ReleaseDoubleArrayElements (addMatrixJ, addMatrixH, JNI_ABORT);
	
	return converged;
}


/**
 * Cuda processing, using single precision for speed<br>
 * Process {@code fxnMatrix} {@code maximumIterations} times, or until no element changes as much 
 * as {@code epsilon}<br>
 * Processing: {@code (mulMatrix x fxnMatrix) + addMatrix}<br>
 * Input matrices are in row:column order (i.e. laid out with the first being in positions 0 - {@code numCols - 1}
 * 
 * @param fxnMatrixJ	Key matrix
 * @param mulMatrixJ	Square matrix with same number of rows and columns as {@code fxnMatrix} has rows
 * @param addMatrixJ	Matrix with same dimensions as {@code fxnMatrix}
 * @param numRowsJ	Number of rows in {@code fxnMatrix} and {@code addMatrix}.<br>
 * Number of rows and columns in {@code mulMatrix}
 * @param numColsJ	Number of cols in {@code fxnMatrix} and {@code addMatrix}
 * @param maxIterations	Maximum number of iterations to run if don't converge
 * @param epsilonJ	Result is converged if every element of {@code fxnMatrix} changes by less than 
 * {@code epsilon} during one round of processing
 * @param whichGPUJ	Which GPU to target
 * @return	The transpose of the final matrix, whether or not it converged, as a 1-d matrix
 */
JNIEXPORT jboolean JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_convergeMatrixCuda (
	JNIEnv *env, jclass clazz, jfloatArray fxnMatrixJ, jfloatArray mulMatrixJ, jfloatArray addMatrixJ, 
	jfloatArray resultsJ, jint numRowsJ, jint numColsJ, jint maxIterations, jint whichGPUJ, jfloat epsilonJ)
{
	int	whichGPU = (int) whichGPUJ;
	cSetDevice (whichGPU, __LINE__);	// Assign code to the appropriate GPU
	
	int		numRows = (int) numRowsJ;
	int		numCols = (int) numColsJ;
	int		maxIter = (int) maxIterations;
	float	epsilon = (float) epsilonJ;
	float	*fxnMatrixH = env->GetFloatArrayElements (fxnMatrixJ, NULL);
	float	*mulMatrixH = env->GetFloatArrayElements (mulMatrixJ, NULL);
	float	*addMatrixH = env->GetFloatArrayElements (addMatrixJ, NULL);
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(float);
	size_t	memSizeMul = numRows * numRows * sizeof(float);
	float	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
	size_t	changeCountH;
	size_t	*changeCount;
	jboolean	converged = false;
	
//	struct timeval startTV;
//	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	// Setup execution parameters
	dim3	threads (kBlockSize, kBlockSize);
	dim3	grid ((numCols + kBlockRounding) / kBlockSize, (numRows + kBlockRounding) / kBlockSize);
	int		i, numBlocks = (arraySize + kBigBlockRounding) / kBigBlockSize;
	
	for (i = 0; i < maxIter; ++i)
	{
		matrixMulCUDA<kBlockSize, float><<< grid, threads >>>(resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows);
		hipDeviceSynchronize ();
		swap(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<kBigBlockSize, float><<< numBlocks, kBigBlockSize >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
	}
	
//	struct timeval endTV;
//	gettimeofday (&endTV, NULL);
//	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
//	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
//	printf ("matrixMulCUDA (float) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	float	*results = env->GetFloatArrayElements (resultsJ, NULL);
	transposeRC<kBlockSize, float, float><<< grid, threads >>>(fxnMatrix, resultHold, numRows, numCols);
	cCopy (results, resultHold, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	env->ReleaseFloatArrayElements (resultsJ, results, 0);	// Copy these results back
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	env->ReleaseFloatArrayElements (fxnMatrixJ, fxnMatrixH, JNI_ABORT);
	env->ReleaseFloatArrayElements (mulMatrixJ, mulMatrixH, JNI_ABORT);
	env->ReleaseFloatArrayElements (addMatrixJ, addMatrixH, JNI_ABORT);
	
	return converged;
}


/**
 * Cuda processing, using double precision rather than single precision<br>
 * Process {@code fxnMatrix} {@code maximumIterations} times, or until no element changes as much 
 * as {@code epsilon}<br>
 * Processing: {@code (mulMatrix x fxnMatrix) + addMatrix}<br>
 * Input matrices are in row:column order (i.e. laid out with the first being in positions 0 - {@code numCols - 1}
 * 
 * @param fxnMatrixJ	Key matrix
 * @param mulMatrixJ	Square matrix with same number of rows and columns as {@code fxnMatrix} has rows
 * @param addMatrixJ	Matrix with same dimensions as {@code fxnMatrix}
 * @param numRowsJ	Number of rows in {@code fxnMatrix} and {@code addMatrix}.<br>
 * Number of rows and columns in {@code mulMatrix}
 * @param numColsJ	Number of cols in {@code fxnMatrix} and {@code addMatrix}
 * @param maxIterations	Maximum number of iterations to run if don't converge
 * @param epsilonJ	Result is converged if every element of {@code fxnMatrix} changes by less than 
 * {@code epsilon} during one round of processing
 * @param whichGPUJ	Which GPU to target
 * @return	The transpose of the final matrix, whether or not it converged, as a 1-d matrix
 */
JNIEXPORT jboolean JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_convergeMatrixCudaD (
	JNIEnv *env, jclass clazz, jdoubleArray fxnMatrixJ, jdoubleArray mulMatrixJ, jdoubleArray addMatrixJ, 
	jdoubleArray resultsJ, jint numRowsJ, jint numColsJ, jint maxIterations, jint whichGPUJ, jdouble epsilonJ)
{
	int	whichGPU = (int) whichGPUJ;
	cSetDevice (whichGPU, __LINE__);	// Assign code to the appropriate GPU
	
	int		numRows = (int) numRowsJ;
	int		numCols = (int) numColsJ;
	int		maxIter = (int) maxIterations;
	double	epsilon = (double) epsilonJ;
	double	*fxnMatrixH = env->GetDoubleArrayElements (fxnMatrixJ, NULL);
	double	*mulMatrixH = env->GetDoubleArrayElements (mulMatrixJ, NULL);
	double	*addMatrixH = env->GetDoubleArrayElements (addMatrixJ, NULL);
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(double);
	size_t	memSizeMul = numRows * numRows * sizeof(double);
	double	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
//	size_t	changeCountTotal = 0;
	size_t	changeCountH;
	size_t	*changeCount;
	jboolean	converged = false;
	
#ifdef DoPrint
	printf ("In convergeMatrixCudaD, numRows: %d, numCols: %d, maxIter: %d\n", numRows, numCols, maxIter);
	printArray ("mulMatrix", mulMatrixH, numRows * numRows);
	printArray ("fxnMatrix", fxnMatrixH, arraySize);
	printArray ("addMatrix", addMatrixH, arraySize);
#endif
	
//	struct timeval startTV;
//	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	// Setup execution parameters
	dim3	threads (kBlockSize, kBlockSize);
	dim3	grid ((numCols + kBlockRounding) / kBlockSize, (numRows + kBlockRounding) / kBlockSize);
	int		i, numBlocks = (arraySize + kBigBlockRounding) / kBigBlockSize;
	
	for (i = 0; i < maxIter; ++i)
	{
#ifdef DoPrint
		printArray ("first", first, arraySize);
#endif
		matrixMulCUDA<kBlockSize, double><<< grid, threads >>>(resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows);
		hipDeviceSynchronize ();
		swapD(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<kBigBlockSize, double><<< numBlocks, kBigBlockSize >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
//		else
//			changeCountTotal += changeCountH;
	}
	
//	struct timeval endTV;
//	gettimeofday (&endTV, NULL);
//	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
//	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
//	printf ("matrixMulCUDA (double) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	double	*results = env->GetDoubleArrayElements (resultsJ, NULL);
	transposeRC<kBlockSize, double, double><<< grid, threads >>>(fxnMatrix, resultHold, numRows, numCols);
	cCopy (results, resultHold, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	env->ReleaseDoubleArrayElements (resultsJ, results, 0);	// Copy these results back
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	env->ReleaseDoubleArrayElements (fxnMatrixJ, fxnMatrixH, JNI_ABORT);
	env->ReleaseDoubleArrayElements (mulMatrixJ, mulMatrixH, JNI_ABORT);
	env->ReleaseDoubleArrayElements (addMatrixJ, addMatrixH, JNI_ABORT);
	
	return converged;
}


/**
 * Cuda processing, using single precision for speed<br>
 * Process {@code fxnMatrix} {@code maximumIterations} times, or until no element changes as much 
 * as {@code epsilon}<br>
 * Processing: {@code (mulMatrix x fxnMatrix) + addMatrix}<br>
 * Input matrices are in row:column order (i.e. laid out with the first being in positions 0 - {@code numCols - 1}
 * 
 * @param fxnMatrixJ	Key matrix
 * @param mulMatrixJ	Square matrix with same number of rows and columns as {@code fxnMatrix} has rows
 * @param addMatrixJ	Matrix with same dimensions as {@code fxnMatrix}
 * @param numRowsJ	Number of rows in {@code fxnMatrix} and {@code addMatrix}.<br>
 * Number of rows and columns in {@code mulMatrix}
 * @param numColsJ	Number of cols in {@code fxnMatrix} and {@code addMatrix}
 * @param maxIterations	Maximum number of iterations to run if don't converge
 * @param epsilonJ	Result is converged if every element of {@code fxnMatrix} changes by less than 
 * {@code epsilon} during one round of processing
 * @param whichGPUJ	Which GPU to target
 * @return	The transpose of the final matrix, (i.e. in column:row order) whether or not it converged, as a 1-d matrix
 */
JNIEXPORT jboolean JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_convergeMatrixCudaOld (
	JNIEnv *env, jclass clazz, jfloatArray fxnMatrixJ, jfloatArray mulMatrixJ, jfloatArray addMatrixJ, 
	jfloatArray resultsJ, jint numRowsJ, jint numColsJ, jint maxIterations, jint whichGPUJ, jfloat epsilonJ)
{
	int	whichGPU = (int) whichGPUJ;
	cSetDevice (whichGPU, __LINE__);	// Assign code to the appropriate GPU
	
	int		numRows = (int) numRowsJ;
	int		numCols = (int) numColsJ;
	int		maxIter = (int) maxIterations;
	float	epsilon = (float) epsilonJ;
	float	*fxnMatrixH = env->GetFloatArrayElements (fxnMatrixJ, NULL);
	float	*mulMatrixH = env->GetFloatArrayElements (mulMatrixJ, NULL);
	float	*addMatrixH = env->GetFloatArrayElements (addMatrixJ, NULL);
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(float);
	size_t	memSizeMul = numRows * numRows * sizeof(float);
	float	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
//	size_t	changeCountTotal = 0;
	size_t	changeCountH;
	size_t	*changeCount;
	jboolean	converged = false;
	
#ifdef DoPrint
	printf ("In convergeMatrixCuda, numRows: %d, numCols: %d, maxIter: %d\n", numRows, numCols, maxIter);
	printArray ("mulMatrix", mulMatrixH, numRows * numRows);
	printArray ("fxnMatrix", fxnMatrixH, arraySize);
	printArray ("addMatrix", addMatrixH, arraySize);
#endif
	
	struct timeval startTV;
	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	// Setup execution parameters
	dim3	threads (kBlockSize, kBlockSize);
	dim3	grid ((numCols + kBlockRounding) / kBlockSize, (numRows + kBlockRounding) / kBlockSize);
	int		i, numBlocks = (arraySize + kBigBlockRounding) / kBigBlockSize;
	
//	struct timeval	midTV;
//	gettimeofday (&midTV, NULL);
	for (i = 0; i < maxIter; ++i)
	{
#ifdef DoPrint
		printArray ("first", first, arraySize);
#endif
		matrixMulCUDA<kBlockSize, float><<< grid, threads >>>(resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows);
		hipDeviceSynchronize ();
		swap(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<kBigBlockSize, float><<< numBlocks, kBigBlockSize >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
//		else
//			changeCountTotal += changeCountH;
	}
	
	struct timeval endTV;
	gettimeofday (&endTV, NULL);
	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
//	unsigned long	theMid = (1000000 * midTV.tv_sec) + midTV.tv_usec;
	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
	printf ("matrixMulCUDA (float) used %lu microseconds for %d iterations\n", done - theStart, i);
//	printf ("Just matrixMulCUDA used %lu microseconds\n", done - theMid);
//	printf ("changeCountTotal = %lu\n", changeCountTotal);
	
	float	*results = env->GetFloatArrayElements (resultsJ, NULL);
//	cCopy (fxnMatrixH, fxnMatrix, memSizeFxn, hipMemcpyDeviceToHost, "(results, result)", __LINE__);
//	transpose<float> (fxnMatrixH, results, numRows, numCols);
	transposeRC<kBlockSize, float, float><<< grid, threads >>>(fxnMatrix, resultHold, numRows, numCols);
	cCopy (results, resultHold, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	env->ReleaseFloatArrayElements (resultsJ, results, 0);	// Copy these results back
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	env->ReleaseFloatArrayElements (fxnMatrixJ, fxnMatrixH, JNI_ABORT);
	env->ReleaseFloatArrayElements (mulMatrixJ, mulMatrixH, JNI_ABORT);
	env->ReleaseFloatArrayElements (addMatrixJ, addMatrixH, JNI_ABORT);
	
	return converged;
}


/**
 * Cuda processing, using double precision rather than single precision<br>
 * Process {@code fxnMatrix} {@code maximumIterations} times, or until no element changes as much 
 * as {@code epsilon}<br>
 * Processing: {@code (mulMatrix x fxnMatrix) + addMatrix}<br>
 * Input matrices are in row:column order (i.e. laid out with the first being in positions 0 - {@code numCols - 1}
 * 
 * @param fxnMatrixJ	Key matrix
 * @param mulMatrixJ	Square matrix with same number of rows and columns as {@code fxnMatrix} has rows
 * @param addMatrixJ	Matrix with same dimensions as {@code fxnMatrix}
 * @param numRowsJ	Number of rows in {@code fxnMatrix} and {@code addMatrix}.<br>
 * Number of rows and columns in {@code mulMatrix}
 * @param numColsJ	Number of cols in {@code fxnMatrix} and {@code addMatrix}
 * @param maxIterations	Maximum number of iterations to run if don't converge
 * @param epsilonJ	Result is converged if every element of {@code fxnMatrix} changes by less than 
 * {@code epsilon} during one round of processing
 * @param whichGPUJ	Which GPU to target
 * @return	The transpose of the final matrix, (i.e. in column:row order) whether or not it converged, as a 1-d matrix
 */
JNIEXPORT jboolean JNICALL Java_CRC_1Prediction_InferenceAlgoCancer_convergeMatrixCudaDOld (
	JNIEnv *env, jclass clazz, jdoubleArray fxnMatrixJ, jdoubleArray mulMatrixJ, jdoubleArray addMatrixJ, 
	jdoubleArray resultsJ, jint numRowsJ, jint numColsJ, jint maxIterations, jint whichGPUJ, jdouble epsilonJ)
{
	int	whichGPU = (int) whichGPUJ;
	cSetDevice (whichGPU, __LINE__);	// Assign code to the appropriate GPU
	
	int		numRows = (int) numRowsJ;
	int		numCols = (int) numColsJ;
	int		maxIter = (int) maxIterations;
	double	epsilon = (double) epsilonJ;
	double	*fxnMatrixH = env->GetDoubleArrayElements (fxnMatrixJ, NULL);
	double	*mulMatrixH = env->GetDoubleArrayElements (mulMatrixJ, NULL);
	double	*addMatrixH = env->GetDoubleArrayElements (addMatrixJ, NULL);
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(double);
	size_t	memSizeMul = numRows * numRows * sizeof(double);
	double	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
//	size_t	changeCountTotal = 0;
	size_t	changeCountH;
	size_t	*changeCount;
	jboolean	converged = false;
	
#ifdef DoPrint
	printf ("In convergeMatrixCuda, numRows: %d, numCols: %d, maxIter: %d\n", numRows, numCols, maxIter);
	printArray ("mulMatrix", mulMatrixH, numRows * numRows);
	printArray ("fxnMatrix", fxnMatrixH, arraySize);
	printArray ("addMatrix", addMatrixH, arraySize);
#endif
	
	struct timeval startTV;
	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	// Setup execution parameters
	dim3	threads (kBlockSize, kBlockSize);
	dim3	grid ((numCols + kBlockRounding) / kBlockSize, (numRows + kBlockRounding) / kBlockSize);
	int		i, numBlocks = (arraySize + kBigBlockRounding) / kBigBlockSize;
	
//	struct timeval	midTV;
//	gettimeofday (&midTV, NULL);
	for (i = 0; i < maxIter; ++i)
	{
#ifdef DoPrint
		printArray ("first", first, arraySize);
#endif
		matrixMulCUDA<kBlockSize, double><<< grid, threads >>>(resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows);
		hipDeviceSynchronize ();
		swapD(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<kBigBlockSize, double><<< numBlocks, kBigBlockSize >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
//		else
//			changeCountTotal += changeCountH;
	}
	
	struct timeval endTV;
	gettimeofday (&endTV, NULL);
	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
//	unsigned long	theMid = (1000000 * midTV.tv_sec) + midTV.tv_usec;
	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
	printf ("matrixMulCUDA (double) used %lu microseconds for %d iterations\n", done - theStart, i);
//	printf ("Just matrixMulCUDA used %lu microseconds\n", done - theMid);
//	printf ("changeCountTotal = %lu\n", changeCountTotal);
	
	double	*results = env->GetDoubleArrayElements (resultsJ, NULL);
//	cCopy (fxnMatrixH, fxnMatrix, memSizeFxn, hipMemcpyDeviceToHost, "(results, result)", __LINE__);
//	transpose<double> (fxnMatrixH, results, numRows, numCols);
	transposeRC<kBlockSize, double, double><<< grid, threads >>>(fxnMatrix, resultHold, numRows, numCols);
	cCopy (results, resultHold, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	env->ReleaseDoubleArrayElements (resultsJ, results, 0);	// Copy these results back
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	env->ReleaseDoubleArrayElements (fxnMatrixJ, fxnMatrixH, JNI_ABORT);
	env->ReleaseDoubleArrayElements (mulMatrixJ, mulMatrixH, JNI_ABORT);
	env->ReleaseDoubleArrayElements (addMatrixJ, addMatrixH, JNI_ABORT);
	
	return converged;
}


