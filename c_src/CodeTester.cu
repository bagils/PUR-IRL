#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <time.h>
#include <cmath>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

using namespace std;

#define swap(a, b) {float *hold = a; a = b; b = hold;}
#define swapD(a, b) {double *hold = a; a = b; b = hold;}
#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif
#define	kBlockSize	32

/**
 * Matrix multiplication and addition on the device: {@code result = (mulMatrix * fxnMatrix) + addMatrix}
 * {@code widthMul} is {@code mulMatrix}'s width and {@code widthFxn} is {@code fxnMatrix}'s width
 * {@code result} and {@code addMatrix} have height {@code heightMul and width {@code widthFxn}
 */
template <int BLOCK_SIZE, class T> __global__ void
matrixMulCUDA (T *result, T *mulMatrix, T *fxnMatrix, T *addMatrix, int widthMul, int heightMul, int widthFxn, int heightFxn)
{
	// Block index
	int		blockX = blockIdx.x * BLOCK_SIZE;
	int		blockY = blockIdx.y * BLOCK_SIZE;
	
	// Thread index
	int		threadX = threadIdx.x;
	int		threadY = threadIdx.y;
	int		maxFxn = widthFxn * heightFxn;
	int		maxMul = widthMul * heightMul;
	bool	deadFxnX, deadMulY;
	bool	deadResX = deadFxnX = (blockX + threadX) >= widthFxn;	// Are these within the valid range of the results matrix?
	bool	deadResY = deadMulY = (blockY + threadY) >= heightMul;
	
	// Offsets from upper left corner to the cells of interest for this thread
	int	mulOffset = (widthMul * threadY) + threadX;
	int	fxnOffset = (widthFxn * threadY) + threadX;
	
	// Index of the first sub-matrix of mulMatrix processed by the block
	int	mulBegin = widthMul * blockY;
	
	// Index of the first sub-matrix of mulMatrix NOT processed by the block
	int	mulEnd = mulBegin + widthMul;
	
	// Step size used to iterate through the sub-matrices of mulMatrix
	int	mulStep = BLOCK_SIZE;
	
	// Index of the first sub-matrix of fxnMatrix processed by the block
	int	fxnBegin = blockX;
	
	// Step size used to iterate through the sub-matrices of fxnMatrix
	int	fxnStep = BLOCK_SIZE * widthFxn;
	
	// resultSub is used to store the element of the block sub-matrix
	// that is computed by the thread
	T	resultSub = 0;
	
	// Loop over all the sub-matrices of mulMatrix and fxnMatrix
	// required to compute the block sub-matrix
	for (int mulUpperLeft = mulBegin, fxnUpperLeft = fxnBegin; mulUpperLeft < mulEnd; mulUpperLeft += mulStep, fxnUpperLeft += fxnStep)
	{
		__shared__ T	mulSub[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ T	fxnSub[BLOCK_SIZE][BLOCK_SIZE];
		
//		printf ("Loading data: Block[%d, %d]: Thread[%d, %d]\n", blockX, blockY, threadX, threadY);
		
		// Load the matrices from device memory to shared memory; each thread loads
		// one element of each matrix.	If block size doesn't evenly divide into array 
		// size, will be attempting to read past the edge.	Make those 0, so nothing 
		// extra gets added
		int mulPos = mulUpperLeft + mulOffset;
		if (!deadMulY && (mulPos < maxMul))
			mulSub[threadY][threadX] = mulMatrix[mulPos];
		else
			mulSub[threadY][threadX] = 0.0f;
		
		int fxnPos = fxnUpperLeft + fxnOffset;
		if (!deadFxnX && (fxnPos < maxFxn))
			fxnSub[threadY][threadX] = fxnMatrix[fxnPos];
		else
			fxnSub[threadY][threadX] = 0.0f;
		
		// Synchronize to make sure the matrices are loaded
		__syncthreads ();
		
//		printf ("Loaded data: Block[%d, %d]: Thread[%d, %d]\n", blockX, blockY, threadX, threadY);
		
		// Multiply the two matrices together; each thread computes one element
		// of the block sub-matrix
#pragma unroll
		
		for (int k = 0; k < BLOCK_SIZE; ++k)
		{
			resultSub += mulSub[threadY][k] * fxnSub[k][threadX];
		}
		
		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of mulMatrix and fxnMatrix in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory; each thread writes one element
	if (!deadResX && !deadResY)
	{
		int	pos = ((blockY + threadY) * widthFxn) + blockX + threadX;	// (numRows * width) + numCols
		result[pos] = resultSub + addMatrix[pos];
	}
}


/**
 * Compare the contents of two T[].  If any value of {@code first} differs from {@code second} 
 * by {@code epsilon} or more, return false.  Else return true
 * 
 * @param first		double[] holding values to test, same length as {@code second}
 * @param second	double[] holding values to test, same length as {@code first}
 * @param size		Length of both {@code first} and {@code second}
 * @param epsilon	Test value.  All matching elements of {@code first} and {@code second} must 
 * differ by less than this
 * @param diffCount	Count of values in {@code first} that differ from {@code second} by {@code epsilon} or more, 
 */
template <int BLOCK_SIZE, class T> __global__ void
compare (T *first, T *second, size_t size, T epsilon, size_t *diffCount)
{
	size_t	blockX = blockIdx.x * BLOCK_SIZE;
	size_t	threadX = threadIdx.x;
	size_t	pos = blockX + threadX;
	
	if (pos < size)
	{
		T	value = first[pos] - second[pos];
		
		if (value < 0)
			value = -value;
		
		if (value > epsilon)
			++(*diffCount);
	}
}


/**
 * Copy the contents of {@code source} into {@code target}
 * 
 * @param source	float[] / double[] to read from.  Must not be null, and length {@code theSize}
 * @param target	float[] / double[] to write to.  Must not be null, and length {@code theSize}
 * @param theSize	Size of both arrays
 */
template <int BLOCK_SIZE, class T> __global__ void
copy (const T *source, T *target, int theSize)
{
	// Block index
	int	blockX = blockIdx.x * BLOCK_SIZE;
	
	// Thread index
	int	threadX = threadIdx.x;
	
	//  Source location
	int	pos = blockX + threadX;
	
	if (pos < theSize)
		target[pos] = source[pos];
}


/**
 * Write the transpose of {@code source}, which is in column:row order (i.e. laid out with the first 
 * column being in positions 0 - {@code numRows - 1} into {@code target} in row:column order
 * 
 * @param source	float[] / double[] to read from.  Must not be null, and length {@code numRows * numCols}
 * @param target	float[] / double[] to write to.  Must not be null, and length {@code numRows * numCols}
 * @param numRows	Number of rows in {@code source}, will be number of cols in {@code target}
 * @param numCols	Number of cols in {@code source}, will be number of rows in {@code target}
 */
template <int BLOCK_SIZE, class T, class U> __global__ void
transposeCR (const T *source, U *target, int numRows, int numCols)
{
	// Block index
	int	blockX = blockIdx.x * BLOCK_SIZE;
	int	blockY = blockIdx.y * BLOCK_SIZE;
	
	// Thread index
	int	threadX = threadIdx.x;
	int	threadY = threadIdx.y;
	
	//  Source location
	int	row = blockY + threadY;
	int	col = blockX + threadX;
	
	if ((row < numRows) && (col < numCols))
	{
		int	readPos = (col * numRows) + row;
		int	writePos = (row * numCols) + col;
		
		target[writePos] = (U) source[readPos];
	}
}


/**
 * Write the transpose of {@code source}, which is in row:column order (i.e. laid out with the first 
 * row being in positions 0 - {@code numCols - 1} into {@code target} in column:row order<br>
 * Runs on the Device, on arrays on the device
 * 
 * @param source	float[] / double[] to read from.  Must not be null, and length {@code numRows * numCols}
 * @param target	float[] / double[] to write to.  Must not be null, and length {@code numRows * numCols}
 * @param numRows	Number of rows in {@code source}, will be number of cols in {@code target}
 * @param numCols	Number of cols in {@code source}, will be number of rows in {@code target}
 */
template <int BLOCK_SIZE, class T, class U> __global__ void
transposeRC (const T *source, U *target, int numRows, int numCols)
{
	// Block index
	int	blockX = blockIdx.x * BLOCK_SIZE;
	int	blockY = blockIdx.y * BLOCK_SIZE;
	
	// Thread index
	int	threadX = threadIdx.x;
	int	threadY = threadIdx.y;
	
	//  Source location
	int	row = blockY + threadY;
	int	col = blockX + threadX;
	
	if ((row < numRows) && (col < numCols))
	{
		int	readPos = (row * numCols) + col;
		int	writePos = (col * numRows) + row;
		
		target[writePos] = (U) (source[readPos]);
	}
}


/**
 * Write the transpose of {@code source} into {@code target}, on the Host
 * 
 * @param source	float / double[] to read from.  Must not be null, and length {@code numRows * numCols}
 * @param target	float / double[] to write to.  Must not be null, and length {@code numRows * numCols}
 * @param numRows	Number of rows in {@code source}, will be number of cols in {@code target}
 * @param numCols	Number of cols in {@code source}, will be number of rows in {@code target}
 */
template <class T, class U> void transposeRCHost (const T source[], U target[], int numRows, int numCols)
{
	int	i, readPos = 0;
	
	for (i = 0; i < numRows; ++i)
	{
		int	j, writePos = i;
		
		for (j = 0; j < numCols; ++j)
		{
			target[writePos] = (U) (source[readPos]);
			++readPos;
			writePos += numRows;
		}
	}
}


/**
 *	Chose which GPU to use, exiting on any error
 */
void cSetDevice (int whichGPU, int line)
{
	hipError_t	error = hipSetDevice (whichGPU);

	if (error != hipSuccess)
	{
		printf ("hipSetDevice (%d) returned error %s (code %d), line (%d)\n", whichGPU, hipGetErrorString (error), error, line);
		exit (EXIT_FAILURE);
	}
//	printf ("hipSetDevice (%d) succeeded\n", whichGPU);
}


/**
 *	Chose which GPU to use, exiting on any error
 */
void cuBlasSetup (hipblasHandle_t *handle, int whichGPU, int line)
{
	cSetDevice (whichGPU, line);
	
	hipblasStatus_t	stat = hipblasCreate (handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("CUBLAS initialization failed, hipblasCreate () returned error %s (code %d), line (%d)\n", _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 * Matrix multiplication and addition on the device: {@code result = (mulMatrix * fxnMatrix) + addMatrix}
 * {@code widthMul} is {@code mulMatrix}'s width and {@code widthFxn} is {@code fxnMatrix}'s width
 * {@code result} and {@code addMatrix} have height {@code heightMul and width {@code widthFxn}
 */
void cuBlasMul (hipblasHandle_t handle, float *result, float *mulMatrix, float *fxnMatrix, 
				float *addMatrix, int colsMul, int rowsMul, int colsFxn, int rowsFxn, int line)
{
	hipblasStatus_t	stat;
	const float		alpha = 1.0f;
	const float		beta = 1.0f;
	int				arraySize = colsFxn * rowsFxn;
	
	// Replace contents of result with addMatrix, so can use the beta add, rather than a separate operation
	copy<1024, float><<< (arraySize + 1023) / 1024, 1024 >>>(addMatrix, result, arraySize);
	hipDeviceSynchronize ();
	
	stat = hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsMul, colsFxn, rowsFxn, &alpha, 
						mulMatrix, rowsMul, fxnMatrix, rowsFxn, &beta, result, rowsMul);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("hipblasSgemm () returned error %s (code %d), line (%d)\n", _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 * Matrix multiplication and addition on the device: {@code result = (mulMatrix * fxnMatrix) + addMatrix}
 * {@code widthMul} is {@code mulMatrix}'s width and {@code widthFxn} is {@code fxnMatrix}'s width
 * {@code result} and {@code addMatrix} have height {@code heightMul and width {@code widthFxn}
 */
void cuBlasMul (hipblasHandle_t handle, double *result, double *mulMatrix, double *fxnMatrix, 
				double *addMatrix, int colsMul, int rowsMul, int colsFxn, int rowsFxn, int line)
{
	hipblasStatus_t	stat;
	const double	alpha = 1.0f;
	const double	beta = 1.0f;
	int				arraySize = colsFxn * rowsFxn;
	
	// Replace contents of result with addMatrix, so can use the beta add, rather than a separate operation
	copy<1024, double><<< (arraySize + 1023) / 1024, 1024 >>>(addMatrix, result, arraySize);
	stat = hipblasDgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowsMul, colsFxn, rowsFxn, &alpha, 
						mulMatrix, rowsMul, fxnMatrix, rowsFxn, &beta, result, rowsMul);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf ("hipblasDgemm () returned error %s (code %d), line (%d)\n", _cudaGetErrorEnum (stat), stat, line);
		exit (EXIT_FAILURE);
	}
}


/**
 *	Allocate cuda memory, exiting on any error
 */
void cMalloc (void **target, size_t theSize, const char *name, int line)
{
	hipError_t	error = hipMalloc (target, theSize);

	if (error != hipSuccess)
	{
		printf ("hipMalloc %s returned error %s (code %d), line (%d)\n", name, hipGetErrorString (error), error, line);
		exit (EXIT_FAILURE);
	}
}


/**
 *	Copy to or from cuda memory, exiting on any error
 */
void cCopy (void *target, const void *source, size_t theSize, enum hipMemcpyKind kind, const char *name, int line)
{
	hipError_t	error = hipMemcpy (target, source, theSize, kind);

	if (error != hipSuccess)
	{
		printf ("hipMemcpy %s returned error %s (code %d), line (%d)\n", name, hipGetErrorString (error), error, line);
		exit (EXIT_FAILURE);
	}
}


hipblasHandle_t *handles;

/**
 *	Routine to let us know the library really is loaded
 */
void initCuda (int numGPUs)
{
	handles = (hipblasHandle_t *) malloc (numGPUs * sizeof(hipblasHandle_t));
	
	for (int whichGPU = 0; whichGPU < numGPUs; ++whichGPU)
	{
		cuBlasSetup (handles + whichGPU, whichGPU, __LINE__);	// Create a hipblasHandle_t for each GPU
	}
}


bool convergeMatrixCudaD (double *fxnMatrixH, double *mulMatrixH, double *addMatrixH, double *results, 
						  int numRows, int numCols, int maxIter, int whichGPU, double epsilon)
{
	cSetDevice (whichGPU, __LINE__);	// Assign code to the appropriate GPU
	
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(double);
	size_t	memSizeMul = numRows * numRows * sizeof(double);
	double	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
	size_t	changeCountH;
	size_t	*changeCount;
	bool	converged = false;
	
	struct timeval startTV;
	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	// Setup execution parameters
	dim3	threads (kBlockSize, kBlockSize);
	dim3	grid ((numCols + kBlockSize - 1) / kBlockSize, (numRows + kBlockSize - 1) / kBlockSize);
	int		i, numBlocks = (arraySize + 1023) / 1024;
	
	for (i = 0; i < maxIter; ++i)
	{
		matrixMulCUDA<kBlockSize, double><<< grid, threads >>>(resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows);
		hipDeviceSynchronize ();
		swapD(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<1024, double><<< numBlocks, 1024 >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
	}
	
	struct timeval endTV;
	gettimeofday (&endTV, NULL);
	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
	printf ("matrixMulCUDA (double) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	transposeRC<kBlockSize, double, double><<< grid, threads >>>(fxnMatrix, resultHold, numRows, numCols);
	cCopy (results, resultHold, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	
	return converged;
}


bool convergeMatrixCuBLASD (double *fxnMatrixH, double *mulMatrixH, double *addMatrixH, double *results, 
						    int numRows, int numCols, int maxIter, int whichGPU, double epsilon)
{
	cSetDevice (whichGPU, __LINE__);	// Assign code to the appropriate GPU
	hipblasHandle_t	handle = handles[whichGPU];
	
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(double);
	size_t	memSizeMul = numRows * numRows * sizeof(double);
	double	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
	size_t	changeCountH;
	size_t	*changeCount;
	bool	converged = false;
	
	struct timeval startTV;
	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	
	int		i, numBlocks = (arraySize + 1023) / 1024;
	
	for (i = 0; i < maxIter; ++i)
	{
		cuBlasMul (handle, resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows, __LINE__);
		hipDeviceSynchronize ();
		swapD(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<1024, double><<< numBlocks, 1024 >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
	}
	
	struct timeval endTV;
	gettimeofday (&endTV, NULL);
	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
	printf ("cuBlasMul (double) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	cCopy (results, fxnMatrix, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	
	return converged;
}


bool convergeMatrixCuda (float *fxnMatrixH, float *mulMatrixH, float *addMatrixH, float *results, 
						 int numRows, int numCols, int maxIter, int whichGPU, float epsilon)
{
	cSetDevice (whichGPU, __LINE__);	// Assign code to the appropriate GPU
	
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(float);
	size_t	memSizeMul = numRows * numRows * sizeof(float);
	float	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
	size_t	changeCountH;
	size_t	*changeCount;
	bool	converged = false;
	
	struct timeval startTV;
	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	// Setup execution parameters
	dim3	threads (kBlockSize, kBlockSize);
	dim3	grid ((numCols + kBlockSize - 1) / kBlockSize, (numRows + kBlockSize - 1) / kBlockSize);
	int		i, numBlocks = (arraySize + 1023) / 1024;
	
	for (i = 0; i < maxIter; ++i)
	{
		matrixMulCUDA<kBlockSize, float><<< grid, threads >>>(resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows);
		hipDeviceSynchronize ();
		swap(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<1024, float><<< numBlocks, 1024 >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
	}
	
	struct timeval endTV;
	gettimeofday (&endTV, NULL);
	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
	printf ("matrixMulCUDA (float) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	transposeRC<kBlockSize, float, float><<< grid, threads >>>(fxnMatrix, resultHold, numRows, numCols);
	cCopy (results, resultHold, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	
	return converged;
}


bool convergeMatrixCuBLAS (float *fxnMatrixH, float *mulMatrixH, float *addMatrixH, float *results, 
						   int numRows, int numCols, int maxIter, int whichGPU, float epsilon)
{
	hipblasHandle_t	handle = handles[whichGPU];
	
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(float);
	size_t	memSizeMul = numRows * numRows * sizeof(float);
	float	*mulMatrix, *fxnMatrix, *addMatrix, *resultHold;
	size_t	changeCountH;
	size_t	*changeCount;
	bool	converged = false;
	
	struct timeval startTV;
	gettimeofday (&startTV, NULL);
	
	// Allocate device memory
	cMalloc ((void **) &mulMatrix, memSizeMul, "mulMatrix", __LINE__);
	cMalloc ((void **) &fxnMatrix, memSizeFxn, "fxnMatrix", __LINE__);
	cMalloc ((void **) &addMatrix, memSizeFxn, "addMatrix", __LINE__);
	cMalloc ((void **) &resultHold, memSizeFxn, "resultHold", __LINE__);
	cMalloc ((void **) &changeCount, sizeof(size_t), "changeCount", __LINE__);
	
	// copy host memory to device
	cCopy (mulMatrix, mulMatrixH, memSizeMul, hipMemcpyHostToDevice, "(mulMatrix, mulMatrixH)", __LINE__);
	cCopy (fxnMatrix, fxnMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(fxnMatrix, fxnMatrixH)", __LINE__);
	cCopy (addMatrix, addMatrixH, memSizeFxn, hipMemcpyHostToDevice, "(addMatrix, addMatrixH)", __LINE__);
	
	
	int		i, numBlocks = (arraySize + 1023) / 1024;
	
	for (i = 0; i < maxIter; ++i)
	{
		cuBlasMul (handle, resultHold, mulMatrix, fxnMatrix, addMatrix, numRows, numRows, numCols, numRows, __LINE__);
		hipDeviceSynchronize ();
		swap(resultHold, fxnMatrix);	// Want fxnMatrix to always hold the final results when we exit
		changeCountH = 0;
		cCopy (changeCount, &changeCountH, sizeof(size_t), hipMemcpyHostToDevice, "(changeCount, changeCountH)", __LINE__);
		compare<1024, float><<< numBlocks, 1024 >>>(fxnMatrix, resultHold, arraySize, epsilon, changeCount);
		hipDeviceSynchronize ();
		cCopy (&changeCountH, changeCount, sizeof(size_t), hipMemcpyDeviceToHost, "(changeCountH, changeCount)", __LINE__);
		if (changeCountH == 0)
		{
			converged = true;
			break;
		}
	}
	
	struct timeval endTV;
	gettimeofday (&endTV, NULL);
	unsigned long	theStart = (1000000 * startTV.tv_sec) + startTV.tv_usec;
	unsigned long	done = (1000000 * endTV.tv_sec) + endTV.tv_usec;
	printf ("cuBlasMul (float) used %lu microseconds for %d iterations\n", done - theStart, i);
	
	cCopy (results, fxnMatrix, memSizeFxn, hipMemcpyDeviceToHost, "(results, resultHold)", __LINE__);
	
	hipFree (mulMatrix);
	hipFree (fxnMatrix);
	hipFree (addMatrix);
	hipFree (resultHold);
	hipFree (changeCount);
	
	return converged;
}


template <class T> void dumpMatrix (T *matrix, const char *title, int numRows, int numCols)
{
	printf ("Row Major %s: ", title);
	
	for (int i = 0; (i < 10) && (i < numCols); ++i)
		printf ("%lf\t", (double) matrix[i]);
	printf ("\n");
	
	for (int i = 0; (i < 10) && (i < numCols); ++i)
		printf ("%lf\t", (double) matrix[i + numRows]);
	printf ("\nCol Major %s: ", title);
	
	for (int i = 0; (i < 10) && (i < numRows); ++i)
		printf ("%lf\t", (double) matrix[i]);
	printf ("\n");
	
	for (int i = 0; (i < 10) && (i < numRows); ++i)
		printf ("%lf\t", (double) matrix[i + numCols]);
	printf ("\n");
}


	/**
	 * Compare results from two different matrix multiplications 
	 * 
	 * @param first		Array that's assumed to have the correct answers
	 * @param second	Array whose values we are testing
	 * @param numRows	Number of rows in {@code first} and {@code second}
	 * @param numCols	GPU which produced the results, which will tell us what method was used
	 * @param whichGPU	GPU which produced the results, which will tell us what method was used
	 * @param maxIter	Maximum number of iterations allowed
	 * @param whichRun	Which run we're executing
	 * @param epsilon	Allowed difference between values
	 */
template <class T> void compareMatrix (T *first, T *second, int numRows, int numCols, int maxIter, int whichGPU, int whichRun, T epsilon)
{
	int	numErrors = 0;
	int	numZero = 0;
	int	numNaN = 0;
	int	numInf = 0;
	int	pos = 0;
	
	for (int i = 0; i < numRows; ++i)
	{
		for (int j = 0; j < numCols; ++j, ++pos)
		{
			T	fValue = first[pos];
			T	sValue = second[pos];
			T	value = fValue - sValue;
			
			if (value < 0.0)
				value = -value;
			
			if (value > epsilon)
				++numErrors;
			else if (fValue == 0.0)
				++numZero;
			else if (isnan (fValue) || isnan (sValue))
				++numNaN;
			else if (isinf (fValue) || isinf (sValue))
				++numInf;
		}
	}
	
	printf ("For run %d, GPU %d, %d max iterations, there were %d zero values, %d NaN, %d Inf, and %d errors out of %d values\nFirst results: ", 
			whichRun, whichGPU, maxIter, numZero, numNaN, numInf, numErrors, pos);
	
	printf ("");
	for (int i = 0; (i < 10) && (i < numCols); ++i)
		printf ("%10.4f\t", first[i + numCols]);
	printf ("\nSecond results: ");
	
	for (int i = 0; (i < 10) && (i < numCols); ++i)
		printf ("%10.4f\t", second[i + numCols]);
	printf ("\n");
}


template <class T> void referenceSolve (T *fxnMatrix, T *mulMatrix, T *addMatrix, T *results, int numRows, int numCols)
{
	int	writePos = 0;
	
	for (int i = 0; i < numRows; ++i)
	{
		T	*mulRow = mulMatrix + (i * numRows);
		
		for (int j = 0; j < numCols; ++j)
		{
			T	value = addMatrix[writePos];
			T	*fxnCol = fxnMatrix + j;
			
			for (int k = 0; k < numRows; ++k)
			{
				value += mulRow[k] * fxnCol[k * numCols];
			}
			
			results[writePos] = value;
			++writePos;
		}
	}
}


/**
 * Done with data in column major order, rather than row major order
 */
template <class T> void referenceSolveCol (T *fxnMatrix, T *mulMatrix, T *addMatrix, T *results, int numRows, int numCols)
{
	int	writePos = 0;
	
	for (int i = 0; i < numCols; ++i)
	{
		T	*fxnCol = fxnMatrix + (i * numRows);
		
		for (int j = 0; j < numRows; ++j)
		{
			T	value = addMatrix[writePos];
			T	*mulRow = mulMatrix + j;
			
			for (int k = 0; k < numRows; ++k)
			{
				value += mulRow[k * numRows] * fxnCol[k];
			}
			
			results[writePos] = value;
			++writePos;
		}
	}
}


void testMulD (int numRows, int numCols, int maxIter, int whichGPU, int whichRun)
{
	double	epsilon = (double) 1e-12;
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(double);
	size_t	memSizeMul = numRows * numRows * sizeof(double);
	double	*mulMatrix, *fxnMatrix, *addMatrix, *results, *results1;
	
	mulMatrix = (double *) malloc (memSizeMul);
	addMatrix = (double *) malloc (memSizeFxn);
	fxnMatrix = (double *) malloc (memSizeFxn);
	results = (double *) malloc (memSizeFxn);
	results1 = (double *) malloc (memSizeFxn);
	
	for (int row = 0; row < numRows; ++row)
	{
		double	rowF = row;
		int		pos = row * numCols;	// Row offset
		
		for (int col = 0; col < numCols; ++col)
		{
			double	value = rowF + col;
			
			addMatrix[pos + col] = fxnMatrix[pos + col] = value;
		}
		
		pos = row * numRows;	// Row offset
		
		for (int col = 0; col < numRows; ++col)
		{
			double	value = rowF + col;
			
			mulMatrix[pos + col] = value;
		}
	}
	
	double	*reference = (double *) malloc (memSizeFxn);
	referenceSolve<double> (fxnMatrix, mulMatrix, addMatrix, results, numRows, numCols);
	transposeRCHost<double, double> (results, reference, numRows, numCols);
	convergeMatrixCudaD (fxnMatrix, mulMatrix, addMatrix, results1, numRows, numCols, 1, whichGPU, epsilon);	// Only iterate once
	convergeMatrixCudaD (fxnMatrix, mulMatrix, addMatrix, results, numRows, numCols, maxIter, whichGPU, epsilon);
	
	double	*mulMatrixT, *fxnMatrixT, *addMatrixT, *resultsBLAS, *resultsBLAS1;
	
	mulMatrixT = (double *) malloc (memSizeMul);
	addMatrixT = (double *) malloc (memSizeFxn);
	fxnMatrixT = (double *) malloc (memSizeFxn);
	resultsBLAS = (double *) malloc (memSizeFxn);
	resultsBLAS1 = (double *) malloc (memSizeFxn);
	
	transposeRCHost<double, double> (fxnMatrix, fxnMatrixT, numRows, numCols);
	transposeRCHost<double, double> (addMatrix, addMatrixT, numRows, numCols);
	transposeRCHost<double, double> (mulMatrix, mulMatrixT, numRows, numRows);
	convergeMatrixCuBLASD (fxnMatrixT, mulMatrixT, addMatrixT, resultsBLAS1, numRows, numCols, 1, whichGPU, epsilon);	// Only iterate once
	convergeMatrixCuBLASD (fxnMatrixT, mulMatrixT, addMatrixT, resultsBLAS, numRows, numCols, maxIter, whichGPU, epsilon);
	
//	dumpMatrix<double> (fxnMatrix, "fxnMatrix", numRows, numCols);
//	dumpMatrix<double> (addMatrix, "addMatrix", numRows, numCols);
//	dumpMatrix<double> (mulMatrix, "mulMatrix", numRows, numRows);
	
	compareMatrix<double> (reference, results1, numRows, numCols, 1, whichGPU, whichRun, epsilon);
	compareMatrix<double> (reference, resultsBLAS1, numRows, numCols, 1, whichGPU, whichRun, epsilon);
	compareMatrix<double> (results, resultsBLAS, numRows, numCols, maxIter, whichGPU, whichRun, epsilon);
	
	free ((void *) addMatrix);
	free ((void *) fxnMatrix);
	free ((void *) mulMatrix);
	free ((void *) results);
	free ((void *) results1);
	free ((void *) addMatrixT);
	free ((void *) fxnMatrixT);
	free ((void *) mulMatrixT);
	free ((void *) resultsBLAS);
	free ((void *) resultsBLAS1);
}


void testMul (int numRows, int numCols, int maxIter, int whichGPU, int whichRun)
{
	float	epsilon = (float) 1e-12;
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(float);
	size_t	memSizeMul = numRows * numRows * sizeof(float);
	float	*mulMatrix, *fxnMatrix, *addMatrix, *results, *results1;
	
	mulMatrix = (float *) malloc (memSizeMul);
	addMatrix = (float *) malloc (memSizeFxn);
	fxnMatrix = (float *) malloc (memSizeFxn);
	results = (float *) malloc (memSizeFxn);
	results1 = (float *) malloc (memSizeFxn);
	
	for (int row = 0; row < numRows; ++row)
	{
		float	rowF = row;
		int		pos = row * numCols;	// Row offset
		
		for (int col = 0; col < numCols; ++col)
		{
			float	value = rowF + col;
			
			addMatrix[pos + col] = fxnMatrix[pos + col] = value;
		}
		
		pos = row * numRows;	// Row offset
		
		for (int col = 0; col < numRows; ++col)
		{
			float	value = rowF + col;
			
			mulMatrix[pos + col] = value;
		}
	}
	
	float	*reference = (float *) malloc (memSizeFxn);
	referenceSolve<float> (fxnMatrix, mulMatrix, addMatrix, results, numRows, numCols);
	transposeRCHost<float, float> (results, reference, numRows, numCols);
	convergeMatrixCuda (fxnMatrix, mulMatrix, addMatrix, results1, numRows, numCols, 1, whichGPU, epsilon);
	convergeMatrixCuda (fxnMatrix, mulMatrix, addMatrix, results, numRows, numCols, maxIter, whichGPU, epsilon);
	
	float	*mulMatrixT, *fxnMatrixT, *addMatrixT, *resultsBLAS, *resultsBLAS1;
	
	mulMatrixT = (float *) malloc (memSizeMul);
	addMatrixT = (float *) malloc (memSizeFxn);
	fxnMatrixT = (float *) malloc (memSizeFxn);
	resultsBLAS = (float *) malloc (memSizeFxn);
	resultsBLAS1 = (float *) malloc (memSizeFxn);
	
	transposeRCHost<float, float> (fxnMatrix, fxnMatrixT, numRows, numCols);
	transposeRCHost<float, float> (addMatrix, addMatrixT, numRows, numCols);
	transposeRCHost<float, float> (mulMatrix, mulMatrixT, numRows, numRows);
	convergeMatrixCuBLAS (fxnMatrixT, mulMatrixT, addMatrixT, resultsBLAS1, numRows, numCols, 1, whichGPU, epsilon);	// Only iterate once
	convergeMatrixCuBLAS (fxnMatrixT, mulMatrixT, addMatrixT, resultsBLAS, numRows, numCols, maxIter, whichGPU, epsilon);
	
//	dumpMatrix<float> (fxnMatrix, "fxnMatrix", numRows, numCols);
//	dumpMatrix<float> (addMatrix, "addMatrix", numRows, numCols);
//	dumpMatrix<float> (mulMatrix, "mulMatrix", numRows, numRows);
	
	compareMatrix<float> (reference, results1, numRows, numCols, 1, whichGPU, whichRun, epsilon);
	compareMatrix<float> (reference, resultsBLAS1, numRows, numCols, 1, whichGPU, whichRun, epsilon);
	compareMatrix<float> (results, resultsBLAS, numRows, numCols, maxIter, whichGPU, whichRun, epsilon);
	
	free ((void *) addMatrix);
	free ((void *) fxnMatrix);
	free ((void *) mulMatrix);
	free ((void *) results);
	free ((void *) results1);
	free ((void *) addMatrixT);
	free ((void *) fxnMatrixT);
	free ((void *) mulMatrixT);
	free ((void *) resultsBLAS);
	free ((void *) resultsBLAS1);
}


void testMulBLAS (int numRows, int numCols, int maxIter, int whichGPU, int whichRun)
{
	float	epsilon = (float) 1e-12;
	size_t	arraySize = numRows * numCols;
	size_t	memSizeFxn = arraySize * sizeof(float);
	size_t	memSizeMul = numRows * numRows * sizeof(float);
	float	*mulMatrix, *fxnMatrix, *addMatrix, *resultsBLAS;
	
	mulMatrix = (float *) malloc (memSizeMul);
	addMatrix = (float *) malloc (memSizeFxn);
	fxnMatrix = (float *) malloc (memSizeFxn);
	resultsBLAS = (float *) malloc (memSizeFxn);
	
	for (int col = 0; col < numCols; ++col)
	{
		float	colF = col;
		int		pos = col * numRows;	// Col offset
		
		for (int row = 0; row < numRows; ++row)
		{
			float	value = colF + row;
			
			addMatrix[pos + row] = fxnMatrix[pos + row] = value;
		}
	}
	
	for (int col = 0; col < numRows; ++col)
	{
		float	colF = col;
		int		pos = col * numRows;	// Col offset
		
		for (int row = 0; row < numRows; ++row)
		{
			float	value = colF + row;
			
			mulMatrix[pos + row] = value;
		}
	}
	
	float	*reference = (float *) malloc (memSizeFxn);
	referenceSolveCol<float> (fxnMatrix, mulMatrix, addMatrix, reference, numRows, numCols);
	convergeMatrixCuBLAS (fxnMatrix, mulMatrix, addMatrix, resultsBLAS, numRows, numCols, maxIter, whichGPU, epsilon);
	
//	dumpMatrix<float> (fxnMatrix, "fxnMatrix", numRows, numCols);
//	dumpMatrix<float> (addMatrix, "addMatrix", numRows, numCols);
//	dumpMatrix<float> (mulMatrix, "mulMatrix", numRows, numRows);
	
	compareMatrix<float> (reference, resultsBLAS, numRows, numCols, maxIter, whichGPU, whichRun, epsilon);
	
	free ((void *) addMatrix);
	free ((void *) fxnMatrix);
	free ((void *) mulMatrix);
	free ((void *) resultsBLAS);
}


int main (int argc, char **argv)
{
	int	numRows = 1758;
	int	numCols = 296;
	int	maxIter = 1000;
	int	numGPUs = 4;
	int	numRuns = 1;
	
	initCuda (numGPUs);
	for (int run = 0; run < numRuns; ++run)
	{
		for (int whichGPU = 0; whichGPU < numGPUs; ++whichGPU)
		{
			testMulD (numRows, numCols, maxIter, whichGPU, run);
//			testMul (numRows, numCols, maxIter, whichGPU, run);
//			testMulBLAS (numRows, numCols, maxIter, whichGPU, run);
		}
	}
}
